#include "hip/hip_runtime.h"
//
// Created by Matteo on 25/01/2020.
//

#include <iostream>
#include <chrono>
#include "processing.h"

using namespace std;
using namespace chrono;

// Kernel CUDA function that processes the pixel identified by the expression <blockIdx.x * blockDim.x + threadIdx.x>.
// This process uses a 3x3 kernel matrix composed by 1/9 as kernel values.
// This applies a soft blur to the image.
__global__ void kernel_proc(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    const unsigned int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x; // Pixel index
    int filter_size = 1; // This is set to 1 in order to loop from -filter_size to +filter_size

    // X and Y coordinates of the pixel
    int x = pixel_idx % width;
    int y = (pixel_idx-x) / width;

    // Checking if the index is within the image borders
    if (pixel_idx < width*height) {
        // Accumulators for the kernel processing.
        // Each variable keeps the accumulated value for the multiplication with the original band value by the corresponding value in the kernel.
        float red_acc = 0;
        float green_acc = 0;
        float output_blue = 0;
        // n keeps track of how many multiplications occurs for the current pixel:
        // 3 on the vertices (0,0; 0,img.width; img.height,0; img.height,img.width), 6 on the edges and 9 for the rest of the image.
        int n = 0;
        for (int ox = -filter_size; ox < filter_size+1; ++ox) {
            for (int oy = -filter_size; oy < filter_size+1; ++oy) {
                if ((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) { // Checks image boundaries
                    // Looping on kernel values and storing the accumulator values for the 3 bands
                    const int current_idx = (pixel_idx + ox + oy * width)*3;
                    red_acc += input_image[current_idx];
                    green_acc += input_image[current_idx + 1];
                    output_blue += input_image[current_idx + 2];
                    n++;
                }
            }
        }
        output_image[pixel_idx*3] = red_acc/n;
        output_image[pixel_idx*3+1] = green_acc/n;
        output_image[pixel_idx*3+2] = output_blue/n;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {

    unsigned char* dev_input;
    unsigned char* dev_output;

    hipMalloc((void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMemcpy(dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMalloc((void**) &dev_output, width*height*3*sizeof(unsigned char));

    printf("Processing the image..\n");
    chrono::steady_clock::time_point time_1 = chrono::steady_clock::now();

    dim3 blockDims(512, 1, 1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
    kernel_proc<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);

    chrono::steady_clock::time_point time_2 = steady_clock::now();
    printf("Time elapsed: %Idns\n", chrono::duration_cast<nanoseconds>(time_2 - time_1).count());

    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_output);
}