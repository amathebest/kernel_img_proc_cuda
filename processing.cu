#include "hip/hip_runtime.h"
//
// Created by Matteo on 25/01/2020.
//

#include <iostream>
#include <chrono>
#include "processing.h"

using namespace std;
using namespace chrono;

// Kernel CUDA function that processes the pixel identified by the expression <blockIdx.x * blockDim.x + threadIdx.x>.
// This process uses a 3x3 kernel matrix composed by 1/9 as kernel values.
// This applies a soft blur to the image.
__global__ void kernel_proc(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    const unsigned int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x; // Pixel index
    int filter_size = 1; // This is set to 1 in order to loop from -filter_size to +filter_size

    // X and Y coordinates of the pixel
    int x = pixel_idx % width;
    int y = (pixel_idx-x) / width;

    // Checking if the index is within the image borders
    if (pixel_idx < width*height) {
        // Accumulators for the kernel processing.
        // Each variable keeps the accumulated value for the multiplication with the original band value by the corresponding value in the kernel.
        float red_acc = 0;
        float green_acc = 0;
        float output_blue = 0;
        // n keeps track of how many multiplications occurs for the current pixel:
        // 3 on the vertices (0,0; 0,img.width; img.height,0; img.height,img.width), 6 on the edges and 9 for the rest of the image.
        int n = 0;
        for (int krow = -filter_size; krow < filter_size+1; krow++) {
            for (int kcol = -filter_size; kcol < filter_size+1; kcol++) {
                if ((x + krow) > -1 && (x + krow) < width && (y + kcol) > -1 && (y + kcol) < height) { // Checks image boundaries
                    // Looping on kernel values and storing the accumulator values for the 3 bands
                    const int current_idx = (pixel_idx + krow + kcol * width)*3;
                    red_acc += input_image[current_idx];
                    green_acc += input_image[current_idx + 1];
                    output_blue += input_image[current_idx + 2];
                    n++;
                }
            }
        }
        output_image[pixel_idx*3] = red_acc/n;
        output_image[pixel_idx*3+1] = green_acc/n;
        output_image[pixel_idx*3+2] = output_blue/n;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {
    int array_size = width*height*3;
    unsigned char* dev_input;
    unsigned char* dev_output;

    hipMalloc((void**) &dev_input, array_size*sizeof(unsigned char));
    hipMemcpy(dev_input, input_image, array_size*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMalloc((void**) &dev_output, array_size*sizeof(unsigned char));

    printf("Processing the image..\n");
    chrono::steady_clock::time_point time_1 = chrono::steady_clock::now();

    dim3 blockDims(512, 1, 1);
    dim3 gridDims((unsigned int) ceil((double)(array_size/blockDims.x)), 1, 1);
    kernel_proc<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);

    chrono::steady_clock::time_point time_2 = steady_clock::now();
    printf("Time elapsed: %Id[ns]\n", chrono::duration_cast<nanoseconds>(time_2 - time_1).count());

    hipMemcpy(output_image, dev_output, array_size*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_output);
}