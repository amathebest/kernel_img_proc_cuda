#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "lodepng.h"
#include "lodepng.cpp"
#include "utils.h"

using namespace std;
using namespace lodepng;
using namespace chrono;


__global__ void kernel_proc(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    const unsigned int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = pixel_idx % width;
    int y = (pixel_idx-x)/width;
    int filter_size = 1; // Size of the filter goes from -1 to 1
    // Checking if the index is within the image borders
    if (pixel_idx < width*height) {
        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for (int ox = -filter_size; ox < filter_size+1; ++ox) {
            for (int oy = -filter_size; oy < filter_size+1; ++oy) {
                if ((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (pixel_idx+ox+oy*width)*3;
                    output_red += input_image[currentoffset];
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[pixel_idx*3] = output_red/hits;
        output_image[pixel_idx*3+1] = output_green/hits;
        output_image[pixel_idx*3+2] = output_blue/hits;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {

    unsigned char* dev_input;
    unsigned char* dev_output;

    hipMalloc((void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMemcpy(dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMalloc((void**) &dev_output, width*height*3*sizeof(unsigned char));

    printf("Processing the image..\n");
    steady_clock::time_point time_1 = steady_clock::now();

    dim3 blockDims(512, 1, 1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
    kernel_proc<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);

    steady_clock::time_point time_2 = steady_clock::now();
    printf("Time elapsed: %Idns\n", duration_cast<nanoseconds>(time_2 - time_1).count());

    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_output);
}

int main() {
    // Base path for input and output folders
    string base_path = "C:\\Users\\Matteo\\Dropbox\\University\\11\\PC\\kernel_img_proc_cuda";
    string input_folder = base_path + "\\input\\";
    string output_folder = base_path + "\\output\\";
    string file_name = "pic.png";
    string file_in = input_folder + file_name;
    string file_out = output_folder + file_name;

    // Image declaration and reading
    vector<unsigned char> img;
    unsigned int width, height;
    unsigned error = decode(img, width, height, file_in);

    unsigned char* input_image = new unsigned char[(img.size()*3)/4];
    unsigned char* output_image = new unsigned char[(img.size()*3)/4];

    int pixel_idx = 0;
    for (int i = 0; i < img.size(); ++i) {
        if ((i+1) % 4 != 0) { // Skipping transparency
            input_image[pixel_idx] = img.at(i);
            output_image[pixel_idx] = 255;
            pixel_idx++;
        }
    }

    // Applying blur effect
    filter(input_image, output_image, width, height);

    // Prepare data for output
    vector<unsigned char> out_image;
    for (int i = 0; i < img.size(); ++i) {
        out_image.push_back(output_image[i]);
        if ((i+1) % 3 == 0) { // Re-applying transparency
            out_image.push_back(255);
        }
    }

    // Picture output
    error = encode(file_out, out_image, width, height);

    return 0;
}
